#include "hip/hip_runtime.h"
#include "dummy.hh"

#include <stdio.h>
#include <iostream>

__global__ void dummy (int* v)
{
  // Calling a device function in a device function
  // fails with: "nvlink error: Undefined reference to ..."
  // if not compiled with the appropriate commands.
  v[0] = 0;
  foo (v);
  printf ("%i\n", v[0]);
}

int main()
{
  int* v;
  hipMalloc (&v, 4 * sizeof (int));
  dummy<<<1,1>>> (v);
  hipFree (v);

  int* u;
  u = (int*) malloc (4 * sizeof (int));

  // Calling the following host function works as expected
  bar (u);
  std::cout << u[0] << std::endl;

  // Calling a device function in a host function fails
  // as expected (detected by the compiler):
  // error: calling a __device__ function("foo") from a
  // __host__ function("main") is not allowed
  //foo (u);

  free (u);

  return 0;
}
